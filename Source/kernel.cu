#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.h"

#define L_Matrix(row,col) matL[((row)*numRows + (col))]
#define L_Matrix_mat(row,col) matrix[((row)*numRows + (col))]
#define L_Matrix_t(col,row) matL[((row)*numRows + (col))]

#define FATAL(msg, ...) \
    do {\
        fprintf(stderr, "[%s:%d] "msg"\n", __FILE__, __LINE__, ##__VA_ARGS__);\
        exit(-1);\
    } while(0)
//#define DEBUGGING 1



using namespace std;


__global__ void gpu_square_update_kernel_transposed(int* matL, int* vecX, int* vecB, int numRows)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;

	int y = idy * 2;
	int x = idx * 2;
	int top_tri_idx = y;

	if (x == 0)
	{
		vecB[y + 1] = (vecB[y + 1] - L_Matrix_t(top_tri_idx + 1, top_tri_idx) / L_Matrix_t(top_tri_idx, top_tri_idx)*vecB[y]) / L_Matrix_t(top_tri_idx + 1, top_tri_idx + 1);
		vecB[y] = vecB[y] / L_Matrix_t(top_tri_idx, top_tri_idx);
	}

	if (idx >= numRows / 2 || idy >= numRows / 2)
		return;

	if (idy <= idx)
		return;

	//element 1,0 (y,x) (row,col)
	L_Matrix_t(y + 1, x) = (L_Matrix_t(y + 1, x) - L_Matrix_t(top_tri_idx + 1, top_tri_idx) / L_Matrix_t(top_tri_idx, top_tri_idx)*L_Matrix_t(y, x)) / L_Matrix_t(top_tri_idx + 1, top_tri_idx + 1);

	//element 1,1 (y,x) (row,col)
	L_Matrix_t(y + 1, x + 1) = (L_Matrix_t(y + 1, x + 1) - L_Matrix_t(top_tri_idx + 1, top_tri_idx) / L_Matrix_t(top_tri_idx, top_tri_idx)*L_Matrix_t(y, x + 1)) / L_Matrix_t(top_tri_idx + 1, top_tri_idx + 1);

	//element 0,0 (y,x) (row,col)
	L_Matrix_t(y, x) = L_Matrix_t(y, x) / L_Matrix_t(top_tri_idx, top_tri_idx);

	//element 0,1 (y,x) (row,col)
	L_Matrix_t(y, x + 1) = L_Matrix_t(y, x + 1) / L_Matrix_t(top_tri_idx, top_tri_idx);
}

__global__ void gpu_Multiply(int* matL, int* vecX, int* vecB)
{
	//int idx = blockIdx.x*blockDim.x + threadIdx.x;


	for (int i = 0; i < blockDim.x; i++)
	{
		vecB[threadIdx.x] = vecB[threadIdx.x] + (vecX[threadIdx.x] * matL[(threadIdx.x * blockDim.x) + i]);
		//printf(" (%d, %d) \n",((threadIdx.x * blockDim.x)+i), matL[(threadIdx.x * blockDim.x)+i]);
	}


}

__global__ void gpu_simple_solver_kernel(int* matL, int* vecX, int* vecB, int numRows, int i)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx >= numRows)
		return;
	//update the B value for every thread by subtracting off the known x (which was calculating last iteration)
	//multiplied by the corresponding L element
	if (i != 0)
		vecB[idx] = vecB[idx] - matL[(idx*numRows + i) - 1] * vecX[i - 1];

	if (idx == i)
	{
		vecX[i] = vecB[i] / matL[i*numRows + i];
	}
}

__global__ void gpu_simple_solver_Anjum(int* matL, int* vecX, int* vecB, int numRows)
{	__shared__ int ds_X[N];
	__shared__ int ds_matL[N];
	int rs_B;
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (idx >= numRows)		return;
	rs_B=vecB[idx];
	ds_X[threadIdx.x] = vecX[idx];
	
	for (int j = 0; j < numRows; j++)
	{	ds_matL[threadIdx.x]=matL[(idx*numRows + j) ];	}
	__syncthreads();

	//update the B value for every thread by subtracting off the known x (which was calculating last iteration)
	//multiplied by the corresponding L element
	
	for (int j = 0; j < numRows; j++)
	{if (numRows != 0)
		{rs_B = rs_B - ds_matL[j - 1] * ds_X[j - 1];	}
	  if (idx == j)
		{ds_X[j] = rs_B / ds_matL[j];}
	}

	vecX[idx] = ds_X[threadIdx.x];
}


__global__ void gpu_square_solve_kernel_simple(int* matL, int* vecX, int* vecB, int numRows, int i)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	int col_index = i * 2;
	if (col_index >= numRows)
		return;

	int row_index = idx;

	if (row_index < (i + 1) * 2 || row_index >= numRows)
		return;

	int value = matL[(row_index*numRows) + col_index] * vecB[col_index] + matL[(row_index*numRows) + col_index + 1] * vecB[col_index + 1];
	vecB[row_index] = vecB[row_index] - value;

}


void gpu_simple_solver(int* matL, int* vecX, int* vecB, int numRows)
{
	const unsigned int numThreadsPerBlock = N;
	//const unsigned int numBlocks = (numRows - 1) / numThreadsPerBlock + 1;
	const unsigned int numBlocks = 1;
	// Loop Below Executes 8 Times or for each Row of matL
	/*
	for (int i = 0; i < numRows; i++)
	{
		gpu_simple_solver_kernel <<<numBlocks, numThreadsPerBlock >>>(matL, vecX, vecB, numRows, i);
	}
	*/
	gpu_simple_solver_Anjum <<<numBlocks, numThreadsPerBlock >>>(matL, vecX, vecB, numRows);
}

void gpu_complex_solver(int* matL, int* vecX, int* vecB, int numRows)
{
	dim3 dimGrid((numRows / 2 - 1) / 32 + 1, (numRows / 2 - 1) / 32 + 1, 1);
	dim3 dimBlock(32, 32, 1);

	//gpu_square_update_kernel << <dimGrid, dimBlock >> >(matL, vecX, vecB, numRows);

	const unsigned int numThreadsPerBlock = N;
	//const unsigned int numBlocks = (numRows - 1) / numThreadsPerBlock + 1;
	const unsigned int numBlocks = 1;
	// Executed 4 Times Only should 8 times for each Row
	for (int i = 0; i < (numRows / 2); i++)
	{
		gpu_square_solve_kernel_simple <<<numBlocks, numThreadsPerBlock >>>(matL, vecX, vecB, numRows, i);
	}

	//copy B to X for the verification code in main.cu
	hipMemcpy(vecX, vecB, numRows * sizeof(int), hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
}


